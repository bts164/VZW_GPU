
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <limits>

#include <hip/hip_math_constants.h>
#include <nppdefs.h>

#define MAX_ELEMENT_SIZE 9
// store the element in constant memory for fast broadcast to all threads
__constant__ unsigned char element_ptr[MAX_ELEMENT_SIZE*MAX_ELEMENT_SIZE];

template<typename srcT, typename dstT,
         int BLOCK_WIDTH, int BLOCK_HEIGHT,
         int ELEM_SIZE>
__global__
void laplace_morph_kernel(srcT * __restrict__ src,
                          dstT * __restrict__ dst,
                          unsigned char * __restrict__ elem,
                          int width, int height)
{
    // number of halo pixels around this block
    const int NHALO = ELEM_SIZE / 2;

    // round size up to the nearest multiple of 16 to keep
    // memory aligned and avoid bank conflicts
    const int SHMEM_SIZE = 16*((BLOCK_WIDTH+2*NHALO+15)/16);

    // do calculations in floating point to ensure 4-byte memory alignment
    // in shared memory
    __shared__ float sh_src[BLOCK_HEIGHT+2*NHALO][SHMEM_SIZE];

    // global index of the upper left pixel in the cuda block
    int block_i0 = blockIdx.x * blockDim.x;
    int block_j0 = blockIdx.y * blockDim.y;

    // load image and halo pixels into shared_memory
    for (int j = threadIdx.y; j < (BLOCK_HEIGHT + 2*NHALO); j += blockDim.y) {
        int idx_j = block_j0 - NHALO + j;
#pragma unroll
        for (int i = threadIdx.x; i < (BLOCK_WIDTH + 2*NHALO); i += blockDim.x) {
            int idx_i = block_i0 - NHALO + i;
            if (0 <= idx_i && idx_i < width && 0 <= idx_j && idx_j < height) {
                sh_src[j][i] = src[idx_j*width+idx_i];
            } else {
                sh_src[j][i] = HIP_NAN_F;
            }
        }
    }
    __syncthreads();

    int global_i = block_i0 + threadIdx.x;
    int global_j = block_j0 + threadIdx.y;
    if (global_i >= width || global_j >= height)
        return;

    // calculate the min/max over the element
    float min = (NPP_MAXABS_32F/10),
        max = -(NPP_MAXABS_32F/10);
    for (int j = 0; j < ELEM_SIZE; ++j) {
        for (int i = 0; i < ELEM_SIZE; ++i) {
            if (!element_ptr[j*ELEM_SIZE+i])
                continue;
            float val = sh_src[threadIdx.y+j][threadIdx.x+i];
            if (isnan(val)) continue;
            if (val > max) max = val;
            if (val < min) min = val;
        }
    }

    // write the result back to global memory
    float result = 0.5 * (max + min - 2 * sh_src[NHALO+threadIdx.y][NHALO+threadIdx.x]) + 0.5;
    if (result < 0.0f) result = 0.0f;
    if (result > 255.0f) result = 255.0f;
    dst[global_j*width+global_i] = result;

    return;
}

template<int ELEM_SIZE, typename srcT, typename dstT>
inline void call_laplace_morph_kernel(
    srcT *src, dstT *dst,
    unsigned char *elem,
    int width, int height)
{
    hipMemcpyToSymbol(HIP_SYMBOL(element_ptr), elem, ELEM_SIZE);

    // size calculated using CUDA occupancy calculator
    const int BLOCK_WIDTH = 32;
    const int BLOCK_HEIGHT = 12;
    dim3 block_dim(BLOCK_WIDTH, BLOCK_HEIGHT);
    dim3 grid_dim((width + BLOCK_WIDTH - 1) / BLOCK_WIDTH,
                  (height + BLOCK_HEIGHT - 1) / BLOCK_HEIGHT);
    // process Red component
    laplace_morph_kernel<srcT, dstT, BLOCK_WIDTH, BLOCK_HEIGHT, ELEM_SIZE>
        <<<grid_dim, block_dim>>>(src+0*width*height,
                                  dst+0*width*height,
                                  elem, width, height);
    // process Blue component
    laplace_morph_kernel<srcT, dstT, BLOCK_WIDTH, BLOCK_HEIGHT, ELEM_SIZE>
        <<<grid_dim, block_dim>>>(src+1*width*height,
                                  dst+1*width*height,
                                  elem, width, height);
    // process Green component
    laplace_morph_kernel<srcT, dstT, BLOCK_WIDTH, BLOCK_HEIGHT, ELEM_SIZE>
        <<<grid_dim, block_dim>>>(src+2*width*height,
                                  dst+2*width*height,
                                  elem, width, height);
    hipDeviceSynchronize();
    return;
}

template<typename srcT, typename dstT>
void laplace_morph(srcT *src, dstT *dst,
                   unsigned char *elem, int elem_size,
                   int width, int height)
{
    // Hard code calls to template functions with different element sizes.
    // This allows multiple template kernels to be compiled with optimized size
    // of shared memory required for the element size being used hard coded.

    // The down side obviously is that only a small set of sizes is supported

    // The amount of shared memory in this particular kernel is actually so small
    // that this probably does't matter that much in this case, but this is a common
    // optimization technique used
    switch (elem_size)
    {
    case 1:
        call_laplace_morph_kernel<1>(src, dst, elem, width, height);
        break;
    case 3:
        call_laplace_morph_kernel<3>(src, dst, elem, width, height);
        break;
    case 5:
        call_laplace_morph_kernel<5>(src, dst, elem, width, height);
        break;
    case 7:
        call_laplace_morph_kernel<7>(src, dst, elem, width, height);
        break;
    case 9:
        call_laplace_morph_kernel<9>(src, dst, elem, width, height);
        break;
    default:
        std::cerr << "Error: unsupported element size being used for Morphological Laplacian\n";
        break;
    }
}

// template instantiation
template
void laplace_morph(float *src, unsigned char *dst, unsigned char *elem, int elem_size,
                   int width, int height);
