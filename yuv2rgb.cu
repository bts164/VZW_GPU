
#include <hip/hip_runtime.h>
#include <stdio.h>

template<typename srcT, typename dstT>
__global__
void yuv2rgb_kernel(srcT *src, dstT *dst, int width, int height)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= width || j >= height)
        return;

    int yIdx = j * width + i;
    int uvIdx = (j/2) * (width/2) + i/2;
    srcT *Y = src;
    srcT *U = Y + width * height;
    srcT *V = U + width * height / 4;
    dstT *R = dst;
    dstT *G = R + width * height;
    dstT *B = G + width * height;
    R[yIdx] = Y[yIdx] + 1.370705 * (V[uvIdx] - 128.0);
    G[yIdx] = Y[yIdx] - 0.698001 * (V[uvIdx] - 128.0) - 0.337633 * (U[uvIdx] - 128.0);
    B[yIdx] = Y[yIdx] + 1.732446 * (U[uvIdx] - 128.0);
}

template<typename srcT, typename dstT>
int yuv2rgb(srcT *src, dstT *dst, int width, int height)
{
    dim3 blockSize(32, 12);
    dim3 nBlocks((width+blockSize.x-1)/blockSize.x,
                 (height+blockSize.y-1)/blockSize.y);
    yuv2rgb_kernel<<<nBlocks, blockSize>>>(src, dst, width, height);
    hipDeviceSynchronize();
    return 0;
}

template
int yuv2rgb(unsigned char *src, float *dst, int width, int height);
